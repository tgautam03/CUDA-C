
#include <hip/hip_runtime.h>
#define IN_COARSE_TILE_DIM 32
#define OUT_COARSE_TILE_DIM (IN_COARSE_TILE_DIM - 2)

__global__ void coarse_tiled_gpu_stencil(float *in_arr, float *out_arr, int width, int height, int depth)
{
    // Mapping threads to elements of out_arr
    const int k_start = blockIdx.z*OUT_COARSE_TILE_DIM;
    const int j = blockIdx.y*OUT_COARSE_TILE_DIM + threadIdx.y - 1;
    const int i = blockIdx.x*OUT_COARSE_TILE_DIM + threadIdx.x - 1;

    // Shared memory allocation 
    __shared__ float in_prev_sh_tile[IN_COARSE_TILE_DIM][IN_COARSE_TILE_DIM];
    __shared__ float in_curr_sh_tile[IN_COARSE_TILE_DIM][IN_COARSE_TILE_DIM];
    __shared__ float in_next_sh_tile[IN_COARSE_TILE_DIM][IN_COARSE_TILE_DIM];

    // Moving data to shared memory
    if ((i >= 0) && (i < width) && (j >= 0) && (j < height) && (k_start-1 >= 0) && (k_start-1 < depth))
        in_prev_sh_tile[threadIdx.y][threadIdx.x] = in_arr[(k_start-1)*(width*height)+j*(width) + i];
    else
        in_prev_sh_tile[threadIdx.y][threadIdx.x] = 0.0f;

    if ((i >= 0) && (i < width) && (j >= 0) && (j < height) && (k_start >= 0) && (k_start < depth))
        in_curr_sh_tile[threadIdx.y][threadIdx.x] = in_arr[(k_start)*(width*height)+j*(width) + i];
    else
        in_curr_sh_tile[threadIdx.y][threadIdx.x] = 0.0f;

    for (int k = k_start; k < k_start+OUT_COARSE_TILE_DIM; k++)
    {
        if ((i >= 0) && (i < width) && (j >= 0) && (j < height) && (k+1 >= 0) && (k+1 < depth))
            in_next_sh_tile[threadIdx.y][threadIdx.x] = in_arr[(k+1)*(width*height)+j*(width) + i];
        else
            in_next_sh_tile[threadIdx.y][threadIdx.x] = 0.0f;

        // Ensure that all tiles are loaded before proceeding
	    __syncthreads();

        // Stencil computation
        if ((k > 0 && k < depth-1) && (j > 0 && j < height-1) && (i > 0 && i < width-1))
        {
            if ((threadIdx.y > 0 && threadIdx.y < IN_COARSE_TILE_DIM-1) && (threadIdx.x > 0 && threadIdx.x < IN_COARSE_TILE_DIM-1))
            {
                out_arr[k*width*height + j*width + i] = 1*in_curr_sh_tile[threadIdx.y][threadIdx.x]
                                                        + 1*in_curr_sh_tile[threadIdx.y][threadIdx.x-1]
                                                        + 1*in_curr_sh_tile[threadIdx.y][threadIdx.x+1]
                                                        + 1*in_curr_sh_tile[threadIdx.y-1][threadIdx.x]
                                                        + 1*in_curr_sh_tile[threadIdx.y+1][threadIdx.x]
                                                        + 1*in_prev_sh_tile[threadIdx.y][threadIdx.x]
                                                        + 1*in_next_sh_tile[threadIdx.y][threadIdx.x];
            }
        }
        __syncthreads();

        in_prev_sh_tile[threadIdx.y][threadIdx.x] = in_curr_sh_tile[threadIdx.y][threadIdx.x];
        in_curr_sh_tile[threadIdx.y][threadIdx.x] = in_next_sh_tile[threadIdx.y][threadIdx.x];
    }
}            