
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_CHECK(err) {if (err != hipSuccess){printf("%s in %s at line %d \n", hipGetErrorString(err), __FILE__, __LINE__);exit(EXIT_FAILURE);}}

__global__ void c2g_kernel(float* P_in, float* P_out, int N, int M)
{
    // Working on P_out[i,j]
    int i = blockDim.y*blockIdx.y + threadIdx.y;
    int j = blockDim.x*blockIdx.x + threadIdx.x;

    // Parallel execution check
    if (i < N && j < M)
    {
        // Position of red color pixel in P_in
        int idx = (i*M+j)*3;

        // Color 2 Gray conversion
        P_out[i*M+j] = 0.21f*P_in[idx] + 0.71f*P_in[idx+1] + 0.07f*P_in[idx+2];
    }
}

void c2g_gpu(float* P_in, float* P_out, int N, int M)
{
    // Device array pointers
    float* d_P_in;
    float* d_P_out;

    // Device memory allocation
    hipError_t err_in = hipMalloc((void**) &d_P_in, N*M*3*sizeof(float));
    CUDA_CHECK(err_in);

    hipError_t err_out = hipMalloc((void**) &d_P_out, N*M*sizeof(float));
    CUDA_CHECK(err_out);

    // Copying P_in and P_out to device memory
    hipError_t err_in_ = hipMemcpy(d_P_in, P_in, N*M*3*sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err_in_);

    // Kernel execution
    dim3 dim_block(64, 16, 1);
    dim3 dim_grid(ceil(M/64.0), ceil(N/16.0), 1);

    c2g_kernel<<<dim_grid, dim_block>>>(d_P_in, d_P_out, N, M);

    // Copy back results from device to host
    hipError_t err_out_ = hipMemcpy(P_out, d_P_out, N*M*sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK(err_out_);

    // Free device memory
    hipFree(d_P_in);
    hipFree(d_P_out);
}