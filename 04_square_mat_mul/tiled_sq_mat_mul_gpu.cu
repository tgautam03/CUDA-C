
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define TILE_WIDTH 16

#define CUDA_CHECK(err) {if (err != hipSuccess){printf("%s in %s at line %d \n", hipGetErrorString(err), __FILE__, __LINE__);exit(EXIT_FAILURE);}}

__global__ void tiled_sq_mat_mul_kernel(float* A, float* B, float* C, int N)
{
    // Ensure that TILE_WIDTH = BLOCK_SIZE
    assert(TILE_WIDTH == blockDim.x);
    assert(TILE_WIDTH == blockDim.y);
    
    // Ensure N%TILE_WIDTH == 0
    assert(N % TILE_WIDTH == 0);

    // Details regarding this thread
    int by = blockIdx.y;
    int bx = blockIdx.x; 

    int ty = threadIdx.y;
    int tx = threadIdx.x; 

    // Working on C[i,j]
    int i = TILE_WIDTH*by + ty;
    int j = TILE_WIDTH*bx + tx;

    // Allocating shared memory
    __shared__ float sh_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float sh_B[TILE_WIDTH][TILE_WIDTH];

    // Parallel mat mul
    float value = 0;
    for (int phase = 0; phase < N/TILE_WIDTH; phase++)
    {
        // Load Tiles into shared memory
        sh_A[ty][tx] = A[phase*TILE_WIDTH + i*N+tx];
        sh_B[ty][tx] = B[(phase*TILE_WIDTH + ty)*N+j];
        __syncthreads();

        // Dot product
        for (int k = 0; k < TILE_WIDTH; k++)
            value += sh_A[ty][k] * sh_B[k][tx];
        __syncthreads();
    }
    // Assigning calculated value
    C[i*N+j] = value;
}

void tiled_sq_mat_mul_gpu(float* A, float* B, float* C, int N)
{
    // Device array pointers
    float* d_A;
    float* d_B;
    float* d_C;

    // Device memory allocation
    hipError_t err_A = hipMalloc((void**) &d_A, N*N*sizeof(float));
    CUDA_CHECK(err_A);

    hipError_t err_B = hipMalloc((void**) &d_B, N*N*sizeof(float));
    CUDA_CHECK(err_B);

    hipError_t err_C = hipMalloc((void**) &d_C, N*N*sizeof(float));
    CUDA_CHECK(err_C);

    // Copying A and B to device memory
    hipError_t err_A_ = hipMemcpy(d_A, A, N*N*sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err_A_);

    hipError_t err_B_ = hipMemcpy(d_B, B, N*N*sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err_B_);

    // Kernel execution
    dim3 dim_block(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dim_grid(ceil(N/(float)(TILE_WIDTH)), ceil(N/(float)(TILE_WIDTH)), 1);
    tiled_sq_mat_mul_kernel<<<dim_grid, dim_block>>>(d_A, d_B, d_C, N);

    // Copy back results
    hipError_t err_C_ = hipMemcpy(C, d_C, N*N*sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK(err_C_);

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}