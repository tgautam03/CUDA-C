
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_CHECK(err) {if (err != hipSuccess){printf("%s in %s at line %d \n", hipGetErrorString(err), __FILE__, __LINE__);exit(EXIT_FAILURE);}}

__global__ void mat_mul_kernel(float* A, float* B, float* C, int N1, int N2, int N3)
{
    // Working on C[i,j]
    int i = blockDim.y*blockIdx.y + threadIdx.y;
    int j = blockDim.x*blockIdx.x + threadIdx.x;

    // Parallel mat mul
    if (i < N1 && j < N3)
    {
        // Value at C[i,j]
        float value = 0;
        for (int k = 0; k < N2; k++)
        {
            value += A[i*N2+k] * B[k*N3+j];
        }

        // Assigning calculated value
        C[i*N3+j] = value;
    }
}

void mat_mul_gpu(float* A, float* B, float* C, int N1, int N2, int N3)
{
    // Device array pointers
    float* d_A;
    float* d_B;
    float* d_C;

    // Device memory allocation
    hipError_t err_A = hipMalloc((void**) &d_A, N1*N2*sizeof(float));
    CUDA_CHECK(err_A);

    hipError_t err_B = hipMalloc((void**) &d_B, N2*N3*sizeof(float));
    CUDA_CHECK(err_B);

    hipError_t err_C = hipMalloc((void**) &d_C, N1*N3*sizeof(float));
    CUDA_CHECK(err_C);

    // Copying A and B to device memory
    hipError_t err_A_ = hipMemcpy(d_A, A, N1*N2*sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err_A_);

    hipError_t err_B_ = hipMemcpy(d_B, B, N2*N3*sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err_B_);

    // Kernel execution
    dim3 dim_block(32, 32, 1);
    dim3 dim_grid(ceil(N3/32.0), ceil(N1/32.0), 1);
    mat_mul_kernel<<<dim_grid, dim_block>>>(d_A, d_B, d_C, N1, N2, N3);

    // Copy back results
    hipError_t err_C_ = hipMemcpy(C, d_C, N1*N3*sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK(err_C_);

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}