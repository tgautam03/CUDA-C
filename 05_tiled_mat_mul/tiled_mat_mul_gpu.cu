
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define TILE_WIDTH 16

#define CUDA_CHECK(err) {if (err != hipSuccess){printf("%s in %s at line %d \n", hipGetErrorString(err), __FILE__, __LINE__);exit(EXIT_FAILURE);}}

__global__ void tiled_mat_mul_kernel(float* A, float* B, float* C, int N)
{
    
}

void tiled_mat_mul_gpu(float* A, float* B, float* C, int N)
{
    // Device array pointers
    float* d_A;
    float* d_B;
    float* d_C;

    // Device memory allocation
    hipError_t err_A = hipMalloc((void**) &d_A, N*N*sizeof(float));
    CUDA_CHECK(err_A);

    hipError_t err_B = hipMalloc((void**) &d_B, N*N*sizeof(float));
    CUDA_CHECK(err_B);

    hipError_t err_C = hipMalloc((void**) &d_C, N*N*sizeof(float));
    CUDA_CHECK(err_C);

    // Copying A and B to device memory
    hipError_t err_A_ = hipMemcpy(d_A, A, N*N*sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err_A_);

    hipError_t err_B_ = hipMemcpy(d_B, B, N*N*sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err_B_);

    // Kernel execution
    dim3 dim_block(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dim_grid(ceil(N/(float)(TILE_WIDTH)), ceil(N/(float)(TILE_WIDTH)), 1);
    tiled_mat_mul_kernel<<<dim_grid, dim_block>>>(d_A, d_B, d_C, N);

    // Copy back results
    hipError_t err_C_ = hipMemcpy(C, d_C, N*N*sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK(err_C_);

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}