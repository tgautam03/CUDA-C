#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include "sq_mat_mul_cpu.h"
#include "sq_mat_mul_gpu.h"
#include "unco_sq_mat_mul_gpu.h"
#include "../utils.h"

#define MAX_NUM 10 
#define MIN_NUM -10 

int main(int argc, char const *argv[])
{
    int N = 8000;

    // Generate NxN square matrices A and B
    float* A = (float*)malloc(N*N*sizeof(float));
    float* B = (float*)malloc(N*N*sizeof(float));
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            A[i*N+j] = (float)(rand() % (MAX_NUM - MIN_NUM + 1) + MIN_NUM);
            B[i*N+j] = (float)(rand() % (MAX_NUM - MIN_NUM + 1) + MIN_NUM);
        }
    }

    // // Matrix multiplication on a CPU
    // float* C_cpu = (float*)malloc(N*N*sizeof(float));
    // unsigned long long t1_cpu = myCPUTimer();
    // sq_mat_mul_cpu(A, B, C_cpu, N);
    // unsigned long long t2_cpu = myCPUTimer();
    // printf("CPU execution time (N: %d) : %llu microseconds \n", N, t2_cpu-t1_cpu);

    // Coalesced Matrix multiplication on a GPU
    float* C_gpu = (float*)malloc(N*N*sizeof(float));
    unsigned long long t1_gpu = myCPUTimer();
    sq_mat_mul_gpu(A, B, C_gpu, N);
    unsigned long long t2_gpu = myCPUTimer();
    printf("Coalesced GPU execution time (N: %d): %llu microseconds \n", N, t2_gpu-t1_gpu);

    // Uncoalesced Matrix multiplication on a GPU
    float* unco_C_gpu = (float*)malloc(N*N*sizeof(float));
    unsigned long long unco_t1_gpu = myCPUTimer();
    unco_sq_mat_mul_gpu(A, B, unco_C_gpu, N);
    unsigned long long unco_t2_gpu = myCPUTimer();
    printf("Uncoalesced GPU execution time (N: %d): %llu microseconds \n", N, unco_t2_gpu-unco_t1_gpu);

    // Speedup
    printf("\n");
    // printf("Speed-up using Coalesced GPU against CPU (N: %d): %.3f x  \n", N, (double)(t2_cpu-t1_cpu)/(t2_gpu-t1_gpu));
    // printf("Speed-up using Uncoalesced GPU against CPU (N: %d): %.3f x  \n", N, (double)(t2_cpu-t1_cpu)/(unco_t2_gpu-unco_t1_gpu));
    printf("Speed-up using Coalesced GPU against Uncoalesced GPU (N: %d): %.3f x  \n", N, (double)(unco_t2_gpu-unco_t1_gpu)/(t2_gpu-t1_gpu));
    printf("\n");

    // Asserting Results
    printf("Asserting Results... \n");
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            // assert(fabs(C_cpu[i*N+j] - C_gpu[i*N+j]) < 0.00000001);
            assert(fabs(C_gpu[i*N+j] - unco_C_gpu[i*N+j]) < 0.00000001);
        }
    }
    printf("Asserting Passed! \n");

    // Free memory
    free(A);
    free(B);
    // free(C_cpu);
    free(C_gpu);
    free(unco_C_gpu);
    
    return 0;
}
