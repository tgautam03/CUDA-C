
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_CHECK(err) {if (err != hipSuccess){printf("%s in %s at line %d \n", hipGetErrorString(err), __FILE__, __LINE__);exit(EXIT_FAILURE);}}

__global__ void sq_mat_mul_kernel(float* A, float* B, float* C, int N)
{
    // Working on C[i,j]
    int i = blockDim.y*blockIdx.y + threadIdx.y;
    int j = blockDim.x*blockIdx.x + threadIdx.x;

    // Parallel mat mul
    if (i < N && j < N)
    {
        // Value at C[i,j]
        float value = 0;
        for (int k = 0; k < N; k++)
        {
            value += A[i*N+k] * B[k*N+j];
        }

        // Assigning calculated value
        C[i*N+j] = value;
    }
}

void sq_mat_mul_gpu(float* A, float* B, float* C, int N)
{
    // Device array pointers
    float* d_A;
    float* d_B;
    float* d_C;

    // Device memory allocation
    hipError_t err_A = hipMalloc((void**) &d_A, N*N*sizeof(float));
    CUDA_CHECK(err_A);

    hipError_t err_B = hipMalloc((void**) &d_B, N*N*sizeof(float));
    CUDA_CHECK(err_B);

    hipError_t err_C = hipMalloc((void**) &d_C, N*N*sizeof(float));
    CUDA_CHECK(err_C);

    // Copying A and B to device memory
    hipError_t err_A_ = hipMemcpy(d_A, A, N*N*sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err_A_);

    hipError_t err_B_ = hipMemcpy(d_B, B, N*N*sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err_B_);

    // Kernel execution
    dim3 dim_block(32, 32, 1);
    dim3 dim_grid(ceil(N/32.0), ceil(N/32.0), 1);
    sq_mat_mul_kernel<<<dim_grid, dim_block>>>(d_A, d_B, d_C, N);

    // Copy back results
    hipError_t err_C_ = hipMemcpy(C, d_C, N*N*sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK(err_C_);

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}