
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vec_add_kernel(float* A, float* B, float* C, int N)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i < N)
    {
        C[i] = A[i] + B[i] + 0.0f;;
    }
}

void vec_add_gpu(float* A, float* B, float* C, int N)
{
    // Device array pointers
    float* d_A;
    float* d_B;
    float* d_C;

    // Device memory allocation
    hipError_t err_A = hipMalloc((void**) &d_A, N*sizeof(float));
    if (err_A != hipSuccess)
    {
        printf("%s in %s at line %d \n", hipGetErrorString(err_A), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    } 
    hipError_t err_B = hipMalloc((void**) &d_B, N*sizeof(float));
    if (err_B != hipSuccess)
    {
        printf("%s in %s at line %d \n", hipGetErrorString(err_B), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    } 
    hipError_t err_C = hipMalloc((void**) &d_C, N*sizeof(float));
    if (err_C != hipSuccess)
    {
        printf("%s in %s at line %d \n", hipGetErrorString(err_C), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    } 

    // Copying A and B to device memory
    hipError_t err_A_ = hipMemcpy(d_A, A, N*sizeof(float), hipMemcpyHostToDevice);
    if (err_A_ != hipSuccess)
    {
        printf("%s in %s at line %d \n", hipGetErrorString(err_A_), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    } 
    hipError_t err_B_ = hipMemcpy(d_B, B, N*sizeof(float), hipMemcpyHostToDevice);
    if (err_B_ != hipSuccess)
    {
        printf("%s in %s at line %d \n", hipGetErrorString(err_B_), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    } 

    // Kernel execution
    vec_add_kernel<<<ceil(N/256.0), 256>>>(d_A, d_B, d_C, N);

    // Copy back results from device to host
    hipError_t err_C_ = hipMemcpy(C, d_C, N*sizeof(float), hipMemcpyDeviceToHost);
    if (err_C_ != hipSuccess)
    {
        printf("%s in %s at line %d \n", hipGetErrorString(err_C_), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}