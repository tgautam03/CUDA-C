
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_CHECK(err) {if (err != hipSuccess){printf("%s in %s at line %d \n", hipGetErrorString(err), __FILE__, __LINE__);exit(EXIT_FAILURE);}}

__global__ void vec_add_kernel(float* A, float* B, float* C, int N)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i < N)
    {
        C[i] = A[i] + B[i] + 0.0f;;
    }
}

void vec_add_gpu(float* A, float* B, float* C, int N)
{
    // Device array pointers
    float* d_A;
    float* d_B;
    float* d_C;

    // Device memory allocation
    hipError_t err_A = hipMalloc((void**) &d_A, N*sizeof(float));
    CUDA_CHECK(err_A);
    hipError_t err_B = hipMalloc((void**) &d_B, N*sizeof(float));
    CUDA_CHECK(err_B); 
    hipError_t err_C = hipMalloc((void**) &d_C, N*sizeof(float));
    CUDA_CHECK(err_C);

    // Copying A and B to device memory
    hipError_t err_A_ = hipMemcpy(d_A, A, N*sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err_A_);
    hipError_t err_B_ = hipMemcpy(d_B, B, N*sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(err_B_);

    // Kernel execution
    vec_add_kernel<<<ceil(N/256.0), 256>>>(d_A, d_B, d_C, N);

    // Copy back results from device to host
    hipError_t err_C_ = hipMemcpy(C, d_C, N*sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK(err_C_);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}